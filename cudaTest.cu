#include "hip/hip_runtime.h"

#include <iostream>
#include <mex.h>
#include <mpi.h>
#include <hip/hip_runtime_api.h>

#include "matlabUtils.h"
#include "vecbase.h"

__global__ void AplusB(int *ret, const int a, const int b)
{
  ret[threadIdx.x] += a + b + threadIdx.x;
}

void _cuda_mpi_test_1()
{
  std::cout << " MPI Test 1" << std::endl;

  int rank = -1;
  assert(MPI_Comm_rank(MPI_COMM_WORLD, &rank) == MPI_SUCCESS);

  int *ret = 0;
  checkCudaErrors(hipMallocManaged(&ret, 1000*sizeof(int)));
  assert(ret);
  checkCudaErrors(hipMemset(ret, 0, 1000*sizeof(int)));
  checkCudaErrors(hipDeviceSynchronize());
  
  if(rank == 0) {
    AplusB<<<1, 1000>>>(ret, 10, 1000);
    checkCudaErrors(hipDeviceSynchronize());
  }

  assert(MPI_Bcast(ret, 1000, MPI_INT, 0, MPI_COMM_WORLD) == MPI_SUCCESS);

  if(rank != 0) {
    AplusB<<<1, 1000>>>(ret, 10, 1000);
    checkCudaErrors(hipDeviceSynchronize());
  }

  for(int i = 0; i < 10; i++)
    std::cout << " " << i << "  " << ret[i] << std::endl;

  if(ret) { checkCudaErrors(hipFree(ret)); ret = 0; }
  
  return;
}

void _cuda_mpi_test_2()
{
  std::cout << " MPI Test 2" << std::endl;

  int rank = -1;
  assert(MPI_Comm_rank(MPI_COMM_WORLD, &rank) == MPI_SUCCESS);

  int *ret_d = 0;
  checkCudaErrors(hipMalloc(&ret_d, 1000*sizeof(int)));
  assert(ret_d);
  checkCudaErrors(hipMemset(ret_d, 0, 1000*sizeof(int)));
  
  if(rank == 0) 
    AplusB<<<1, 1000>>>(ret_d, 10, 1000);
  
  assert(MPI_Bcast(ret_d, 1000, MPI_INT, 0, MPI_COMM_WORLD) == MPI_SUCCESS);
  
  if(rank != 0)
    AplusB<<<1, 1000>>>(ret_d, 10, 1000);
  
  int *ret_h = new int [1000];
  assert(ret_h);
  checkCudaErrors(hipMemcpy(ret_h, ret_d, 1000*sizeof(int), hipMemcpyDeviceToHost));
  
  if(ret_d) { checkCudaErrors(hipFree(ret_d)); ret_d = 0; }
  
  for(int i = 0; i < 10; i++)
    std::cout << " " << i << "  " << ret_h[i] << std::endl;
  
  if(ret_h) { delete [] ret_h; ret_h = 0; }
  
  return;
}

void _mpi_test_1()
{
  int rank = -1;
  insist(MPI_Comm_rank(MPI_COMM_WORLD, &rank) == MPI_SUCCESS);

  int n_procs = -100;
  insist(MPI_Comm_size(MPI_COMM_WORLD, &n_procs) == MPI_SUCCESS);
  
  Vec<int> i_data(n_procs);
  i_data.zeros();
  
  i_data.show_in_one_line();

  insist(MPI_Allgather(&rank, 1, MPI_INT, i_data, 1, MPI_INT, MPI_COMM_WORLD) == MPI_SUCCESS);

  i_data.show_in_one_line();
}

void cuda_mpi_test()
{
  std::cout << " CUDA MPI Test" << std::endl;

  _cuda_mpi_test_1();

  _cuda_mpi_test_2();
}

